#include "hip/hip_runtime.h"
#include <cassert>
//#define USE_CUDA_DSA 1

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int64_t ELEMENT_RANGES_BLOCKSIZE = 1024;

__global__ void element_ranges_kernel(
    const int64_t* __restrict__ A,
    const int64_t N,
    int64_t* __restrict__ begins,
    int64_t* __restrict__ ends
) {
    __shared__ int64_t values[ELEMENT_RANGES_BLOCKSIZE];
    
    const int64_t idx = threadIdx.x + (blockDim.x - 1) * blockIdx.x;
    int64_t v;
    const bool thread_active = idx < N;
    if (thread_active) {
        v = A[idx];
        values[threadIdx.x] = v;
    }
    __syncthreads();
    
    if (thread_active) {
        if (idx == 0)
            begins[v] = 0;

        if (threadIdx.x != 0) {
            const int64_t prev = values[threadIdx.x - 1];
            if (v != prev) {
                begins[v] = idx;
                ends[prev] = idx;
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor> element_ranges(
    torch::Tensor sorted_values,
    const int64_t ub
) {
    assert(sorted_values.dim() == 1);
    const int64_t N = sorted_values.size(0);
    torch::Tensor begins = torch::full(
        ub, N,
        torch::TensorOptions().dtype(torch::kInt64).device(sorted_values.device())
    );
    torch::Tensor ends = torch::full(
        ub, N,
        torch::TensorOptions().dtype(torch::kInt64).device(sorted_values.device())
    );
    int64_t num_blocks = (N + ELEMENT_RANGES_BLOCKSIZE - 2) / (ELEMENT_RANGES_BLOCKSIZE - 1);
    element_ranges_kernel<<<dim3(num_blocks), dim3(ELEMENT_RANGES_BLOCKSIZE)>>>(
        sorted_values.data<int64_t>(),
        N,
        begins.data<int64_t>(),
        ends.data<int64_t>()
    );
    return {begins, ends};
}

const int64_t PATCH_WA = 512;
const int64_t PATCH_WB = 512;
const int64_t PATCH_H = 24;

template <typename scalar_t>
__global__ void linear_sd_backward_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    //const int64_t* __restrict__ Ai,
    const int64_t* __restrict__ Bi,
    const int64_t* __restrict__ pair_begins,
    const int64_t* __restrict__ pair_ends,
    const int64_t Ad,
    const int64_t Bd,
    const int64_t h,
    float* __restrict__ outputs
) {
    __shared__ float cached_B[PATCH_H * PATCH_WB];

    //assert(blockDim.x == PATCH_WA);
    const int64_t Ax = threadIdx.x + blockIdx.x * blockDim.x;
    const int64_t ybegin = blockIdx.y * PATCH_H;
    const int64_t yrange = min(PATCH_H, h - ybegin);
    //assert(yrange == PATCH_H);

    float cached_Acol[PATCH_H];
    if (Ax < Ad) {
        for (int64_t i = 0; i < PATCH_H; i++)
            cached_Acol[i] = (i < yrange) * A[Ax + (i < yrange) * (i + ybegin) * Ad];
    }

    for (int64_t Bx = 0; Bx < Bd; Bx += PATCH_WB) {
        __syncthreads();
        for (int64_t i = threadIdx.x; i < PATCH_H * PATCH_WB; i += blockDim.x) {
            const int64_t Bcol = Bx + i % PATCH_WB;
            const int64_t Brow = ybegin + i / PATCH_WB;
            if (Bcol < Bd && Brow < h)
                cached_B[i] = B[Bcol + Brow * Bd];
        }
        __syncthreads();

        if (Ax < Ad) {
            const int64_t pair_id = Ax + Ad * (Bx / PATCH_WB);
            //assert(pair_id < N_pairs);
            const int64_t pair_begin = pair_begins[pair_id];
            const int64_t pair_end = pair_ends[pair_id];
            for (int64_t k = pair_begin; k < pair_end; k++) {
                const int64_t Bpi = Bi[k] - Bx;
                //assert(Bpi >= 0);
                //assert(Bpi < PATCH_WB);
                float sum = 0.0;
                for (int64_t i = 0; i < PATCH_H; i++)
                    sum += float(cached_Acol[i]) * float(cached_B[Bpi + i * PATCH_WB]);
                atomicAdd(&outputs[k], sum);
            }
        }
    }
}



torch::Tensor linear_sd_cuda_backward(
    torch::Tensor input,
    torch::Tensor output_grad,
    torch::Tensor di
) {
    assert(input.layout() == torch::kStrided);
    assert(output_grad.layout() == torch::kStrided);
    assert(input.dim() == 2);
    assert(output_grad.dim() == 2);
    assert(di.dim() == 1);
    const int64_t N = di.size(0);
    //torch::Tensor output_indices = di.floor_divide(input.size(0)); //di / input.size(1);
    //assert(output_indices.dtype() == torch::kInt64);
    //torch::Tensor input_indices = di - input.size(0) * output_indices;
    const int64_t Ad = input.size(1);
    const int64_t Bd = output_grad.size(1);
    const int64_t h = input.size(0);
    const int64_t An = (Ad + PATCH_WA - 1) / PATCH_WA;
    const int64_t Bn = (Bd + PATCH_WB - 1) / PATCH_WB;

    const dim3 grid(An, (h + PATCH_H - 1) / PATCH_H);

    torch::Tensor Ai = torch::remainder(di, Ad);
    //assert(torch::all(Ai < Ad).item<bool>());
    torch::Tensor Bi = di.floor_divide(Ad);
    //assert(torch::all(Bi < Bd).item<bool>());
    //torch::Tensor Apx = Ai.floor_divide(PATCH_WA);
    //assert(torch::all(Apx < An).item<bool>());
    torch::Tensor Bpx = Bi.floor_divide(PATCH_WB);
    //assert(torch::all(Bpx < Bn).item<bool>());
    torch::Tensor pair_ids = Ai + Ad * Bpx;
    torch::Tensor pair_perm = torch::argsort(pair_ids, true);
    torch::Tensor inverse_perm = torch::empty_like(pair_perm);
    inverse_perm.index_put_(
        {pair_perm}, 
        torch::arange(
            N,
            torch::TensorOptions()
                .dtype(pair_perm.dtype())
                .device(pair_perm.device())
        )
    );
    pair_ids = pair_ids.index({pair_perm});
    //assert(pair_ids.layout() == torch::kStrided && pair_ids.stride(0) == 1);
    //assert(torch::all(
    //    pair_ids.index({torch::indexing::Slice(0, -1)}) <=
    //    pair_ids.index({torch::indexing::Slice(1, torch::indexing::None)})
    //).item<bool>());
    //Ai = Ai.index({pair_perm});
    Bi = Bi.index({pair_perm});
    
    auto pair_ranges = element_ranges(pair_ids, Ad * Bn);

    torch::Tensor result = torch::zeros(
        N, 
        torch::TensorOptions().dtype(torch::kFloat32).device(output_grad.device())
    );

    AT_DISPATCH_FLOATING_TYPES_AND2(
        torch::ScalarType::Half,
        torch::ScalarType::BFloat16,
        input.type(),
        "linear_sd_cuda_backward",
        ([&] {
            linear_sd_backward_kernel<scalar_t><<<grid, PATCH_WA>>>(
                input.data<scalar_t>(),
                output_grad.data<scalar_t>(),
                //Ai.data<int64_t>(),
                Bi.data<int64_t>(),
                std::get<0>(pair_ranges).data<int64_t>(),
                std::get<1>(pair_ranges).data<int64_t>(),
                Ad,
                Bd,
                h,
                result.data<float>()
            );
        })
    );

    return result.index({inverse_perm});
}


